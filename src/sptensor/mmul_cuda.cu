#include "hip/hip_runtime.h"
/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <pasta.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include "sptensor.h"
#include "mmul_cuda_kernels.h"


int sptCudaSparseTensorMulMatrix(
    sptSemiSparseTensor *Y,
    sptSparseTensor *X,
    const sptMatrix *U,
    sptIndex const mode,
    sptIndex const impl_num,
    sptNnzIndex const smem_size) 
{
    if(mode >= X->nmodes) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "Cuda SpTns * Mtx", "shape mismatch");
    }
    if(X->ndims[mode] != U->nrows) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "Cuda SpTns * Mtx", "shape mismatch");
    }

    int result;
    sptIndex *ind_buf;
    sptIndex m;
    sptNnzIndexVector fiberidx;
    sptTimer timer;
    sptNewTimer(&timer, 0);
    double sort_time, setfiber_time, allocate_time, preprocess_time, copy_time_cpu, copy_time_gpu, comp_time, total_time;

    /* Sort tensor except mode */
    sptStartTimer(timer);
    sptSparseTensorSortIndexAtMode(X, mode, 0);
    sptStopTimer(timer);
    sort_time = sptPrintElapsedTime(timer, "sptSparseTensorSortIndexAtMode");

    /* Set fibers */
    sptStartTimer(timer);
    sptSemiSparseTensorSetFibers(&fiberidx, X, mode);
    sptStopTimer(timer);
    setfiber_time = sptPrintElapsedTime(timer, "sptSparseTensorSetFibers");

    /* Allocate output Y */
    sptStartTimer(timer);
    ind_buf = new sptIndex[X->nmodes * sizeof *ind_buf];
    for(m = 0; m < X->nmodes; ++m) {
        ind_buf[m] = X->ndims[m];
    }
    ind_buf[mode] = U->ncols;
    result = sptNewSemiSparseTensorWithNnz(Y, X->nmodes, mode, ind_buf, fiberidx.len - 1);
    spt_CheckError(result, "Cuda SpTns * Mtx", NULL);
    delete[] ind_buf;
    if(Y->values.stride != U->stride) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "Cuda SpTns * Mtx", "shape mismatch");
    }
    sptStopTimer(timer);
    allocate_time = sptPrintElapsedTime(timer, "sptNewSemiSparseTensorWithNnz");

    preprocess_time = sort_time + setfiber_time + allocate_time;
    printf("[Total preprocess time]: %lf\n", preprocess_time);

    /* Set indices */
    sptStartTimer(timer);
    sptSemiSparseTensorSetIndices(Y, &fiberidx, X);
    sptStopTimer(timer);
    copy_time_cpu = sptPrintElapsedTime(timer, "Copy indices"); 

    sptIndex stride = U->stride;
    double flen = (double)X->nnz / fiberidx.len;
    sptNnzIndex tmp_flen = (fiberidx.data[1] - fiberidx.data[0]) - flen;
    double fvar = tmp_flen * tmp_flen;
    for(sptNnzIndex i=1; i<fiberidx.len - 1; ++i) {
        tmp_flen = (fiberidx.data[i+1] - fiberidx.data[i]) - flen;
        fvar += tmp_flen * tmp_flen;
    }
    tmp_flen = (X->nnz - fiberidx.data[fiberidx.len - 1]) - flen;
    fvar += tmp_flen * tmp_flen;
    fvar = sqrt(fvar);
    printf("nfibs: %zu, flen: %.2f, fvar: %.2f\n", fiberidx.len, flen, fvar);

    sptValue *Y_val = NULL;
    result = hipMalloc((void **) &Y_val, Y->nnz * stride * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    sptValue *X_val = NULL;
    result = hipMalloc((void **) &X_val, X->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    sptIndex *X_inds_m = NULL;
    result = hipMalloc((void **) &X_inds_m, X->nnz * sizeof (sptIndex));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    sptValue *U_val = NULL;
    result = hipMalloc((void **) &U_val, U->nrows * stride * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    sptNnzIndex *fiberidx_val = NULL;
    result = hipMalloc((void **) &fiberidx_val, fiberidx.len * sizeof (sptNnzIndex));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    
    /* Copy data to GPU */
    sptStartTimer(timer);
    hipMemset(Y_val, 0, Y->nnz * stride * sizeof (sptValue));
    hipMemcpy(X_val, X->values.data, X->nnz * sizeof (sptValue), hipMemcpyHostToDevice);
    hipMemcpy(X_inds_m, X->inds[mode].data, X->nnz * sizeof (sptIndex), hipMemcpyHostToDevice);
    hipMemcpy(U_val, U->values, U->nrows * stride * sizeof (sptValue), hipMemcpyHostToDevice);
    hipMemcpy(fiberidx_val, fiberidx.data, fiberidx.len * sizeof (sptNnzIndex), hipMemcpyHostToDevice);
    sptStopTimer(timer);
    copy_time_gpu = sptPrintElapsedTime(timer, "Device copy");

    const sptNnzIndex max_nblocks = 32768;
    const sptNnzIndex max_nthreads_per_block = 256;
    sptNnzIndex max_nthreadsy = 16;

    sptNnzIndex nthreadsx = 1;
    sptNnzIndex nthreadsy = 1;
    sptNnzIndex all_nblocks = 0;
    sptNnzIndex nblocks = 0;

    const char *env_PASTA_TTM_NTHREADS = getenv("PASTA_TTM_NTHREADS");

    switch(impl_num) {
    case 14:
        if(U->ncols <= max_nthreadsy)
            nthreadsx = U->ncols;
        else
            nthreadsx = max_nthreadsy;
        nthreadsy = max_nthreads_per_block / nthreadsx;

        if(Y->nnz < nthreadsy) {
            nthreadsy = Y->nnz;
            nblocks = 1;
        } else {
            all_nblocks = (Y->nnz + nthreadsy -1) / nthreadsy;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }
        }
        break;
    case 15:
        if(U->ncols <= max_nthreadsy)
            nthreadsx = U->ncols;
        else
            nthreadsx = max_nthreadsy;
        nthreadsy = max_nthreads_per_block / nthreadsx;

        if(Y->nnz < nthreadsy) {
            nthreadsy = Y->nnz;
            nblocks = 1;
        } else {
            all_nblocks = (Y->nnz + nthreadsy -1) / nthreadsy;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }
        }
        sptAssert(smem_size >= nthreadsx * nthreadsy * sizeof (sptValue));
        break;
    }
    dim3 dimBlock(nthreadsx, nthreadsy);
    printf("all_nblocks: %lu, nthreadsx: %lu, nthreadsy: %lu\n", all_nblocks, nthreadsx, nthreadsy);

    /* Computation */
    sptStartTimer(timer);

    switch(impl_num) { 
    case 14:  
        printf("[Cuda SpTns * Mtx] spt_TTMRankRBNnzKernel<<<%lu, (%lu, %lu)>>>\n", nblocks, nthreadsx, nthreadsy);
        spt_TTMRankRBNnzKernel<<<nblocks, dimBlock>>>(
            Y_val, stride, Y->nnz,
            X_val, X->nnz, X_inds_m,
            fiberidx_val, fiberidx.len,
            U_val, U->nrows, U->ncols, stride);
        break; 
    case 15:  
        printf("[Cuda SpTns * Mtx] spt_TTMRankRBNnzKernelSM<<<%lu, (%lu, %lu), %lu>>>\n", nblocks, nthreadsx, nthreadsy, smem_size);
        spt_TTMRankRBNnzKernelSM<<<nblocks, dimBlock, smem_size>>>(
            Y_val, stride, Y->nnz,
            X_val, X->nnz, X_inds_m,
            fiberidx_val, fiberidx.len,
            U_val, U->nrows, U->ncols, stride);
        break; 
    }
    result = hipDeviceSynchronize();
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx kernel");

    sptStopTimer(timer);
    comp_time = sptPrintElapsedTime(timer, "Cuda SpTns * Mtx");

    /* Copy back to CPU */
    sptStartTimer(timer);
    hipMemcpy(Y->values.values, Y_val, Y->nnz * stride * sizeof (sptValue), hipMemcpyDeviceToHost);
    sptStopTimer(timer);
    copy_time_gpu += sptPrintElapsedTime(timer, "Device copy back");

    sptFreeTimer(timer);
    result = hipFree(fiberidx_val);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    result = hipFree(U_val);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    result = hipFree(X_inds_m);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    result = hipFree(X_val);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    result = hipFree(Y_val);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Mtx");
    sptFreeNnzIndexVector(&fiberidx);

    total_time = copy_time_cpu + copy_time_gpu + comp_time;
    printf("[Total time]: %lf\n", total_time);
    printf("\n");

    return 0;
}
