#include "hip/hip_runtime.h"
/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <ParTI.h>


__global__ void spt_sMulKernel(
    sptValue *Z_val, 
    const sptValue * __restrict__ X_val, 
    sptNnzIndex X_nnz,
    sptValue a)
{
    sptNnzIndex num_loops_nnz = 1;
    sptNnzIndex const nnz_per_loop = gridDim.x * blockDim.x;
    if(X_nnz > nnz_per_loop) {
        num_loops_nnz = (X_nnz + nnz_per_loop - 1) / nnz_per_loop;
    }

    const sptNnzIndex tidx = threadIdx.x;
    sptNnzIndex x;

    for(sptNnzIndex nl=0; nl<num_loops_nnz; ++nl) {
        x = blockIdx.x * blockDim.x + tidx + nl * nnz_per_loop;
        if(x < X_nnz) {
            Z_val[x] *= a;
        }
        __syncthreads();
    }

}


/**
 * Multiply a sparse tensors with a scalar.
 * @param[out] Z the result of a*X, should be uninitialized
 * @param[in]  a the input scalar
 * @param[in]  X the input X
 */
int sptCudaSparseTensorMulScalar(sptSparseTensor *Z, sptSparseTensor *X, sptValue a)
{
    sptAssert(a != 0.0);
    int result;

    sptTimer timer;
    sptNewTimer(&timer, 0);

    sptStartTimer(timer);
    sptCopySparseTensor(Z, X, 1);
    sptStopTimer(timer);
    sptPrintElapsedTime(timer, "sptCopySparseTensor");

    sptStartTimer(timer);
    sptValue *Z_val = NULL;
    result = hipMalloc((void **) &Z_val, Z->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "CUDA SpTns MulScalar");
    hipMemcpy(Z_val, Z->values.data, Z->nnz * sizeof (sptValue), hipMemcpyHostToDevice);
    sptValue *X_val = NULL;
    result = hipMalloc((void **) &X_val, X->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "CUDA SpTns MulScalar");
    hipMemcpy(X_val, X->values.data, X->nnz * sizeof (sptValue), hipMemcpyHostToDevice);
    sptStopTimer(timer);
    sptPrintElapsedTime(timer, "Device malloc and copy");

    sptStartTimer(timer);

    const sptNnzIndex max_nblocks = 32768;
    const sptNnzIndex max_nthreads_per_block = 256;

    sptNnzIndex nthreadsx = 1;
    sptNnzIndex all_nblocks = 0;
    sptNnzIndex nblocks = 0;

    if(X->nnz < max_nthreads_per_block) {
        nthreadsx = X->nnz;
        nblocks = 1;
    } else {
        nthreadsx = max_nthreads_per_block;
        all_nblocks = (X->nnz + nthreadsx -1) / nthreadsx;
        if(all_nblocks < max_nblocks) {
            nblocks = all_nblocks;
        } else {
            nblocks = max_nblocks;
        }
    }
    dim3 dimBlock(nthreadsx);
    printf("all_nblocks: %lu, nthreadsx: %lu\n", all_nblocks, nthreadsx);

    printf("[CUDA SpTns MulScalar] spt_sMulKernel<<<%lu, (%lu)>>>\n", nblocks, nthreadsx);
    spt_sMulKernel<<<nblocks, dimBlock>>>(Z_val, X_val, X->nnz, a);
    result = hipDeviceSynchronize();
    spt_CheckCudaError(result != 0, "CUDA SpTns MulScalar kernel");

    sptStopTimer(timer);
    sptPrintElapsedTime(timer, "Cpu SpTns MulScalar");
    sptFreeTimer(timer);
    printf("\n");

    hipMemcpy(Z->values.data, Z_val, Z->nnz * sizeof (sptValue), hipMemcpyDeviceToHost);
    result = hipFree(X_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns MulScalar");
    result = hipFree(Z_val);
    spt_CheckCudaError(result != 0, "CUDA SpTns MulScalar");

    return 0;
}
