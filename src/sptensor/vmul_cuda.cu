/*
    This file is part of ParTI!.

    ParTI! is free software: you can redistribute it and/or modify
    it under the terms of the GNU Lesser General Public License as
    published by the Free Software Foundation, either version 3 of
    the License, or (at your option) any later version.

    ParTI! is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Lesser General Public
    License along with ParTI!.
    If not, see <http://www.gnu.org/licenses/>.
*/

#include <pasta.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include "sptensor.h"
#include "vmul_cuda_kernels.h"


int sptCudaSparseTensorMulVector(
    sptSparseTensor *Y,
    sptSparseTensor *X,
    const sptValueVector *V,
    sptIndex const mode,
    sptIndex const impl_num,
    sptNnzIndex const smem_size) 
{

    if(mode >= X->nmodes) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "Cuda SpTns * Vec", "shape mismatch");
    }
    if(X->ndims[mode] != V->len) {
        spt_CheckError(SPTERR_SHAPE_MISMATCH, "Cuda SpTns * Vec", "shape mismatch");
    }

    int result;
    sptIndex *ind_buf;
    sptNnzIndexVector fiberidx;
    sptTimer timer;
    sptNewTimer(&timer, 0);
    double sort_time, setfiber_time, allocate_time, preprocess_time, copy_time_cpu, copy_time_gpu, comp_time, total_time;

    /* Sort tensor except mode */
    sptStartTimer(timer);
    sptSparseTensorSortIndexAtMode(X, mode, 0);
    sptStopTimer(timer);
    sort_time = sptPrintElapsedTime(timer, "sptSparseTensorSortIndexAtMode");

    /* Set fibers */
    sptStartTimer(timer);
    sptSparseTensorSetFibers(&fiberidx, mode, X);
    sptStopTimer(timer);
    setfiber_time = sptPrintElapsedTime(timer, "sptSparseTensorSetFibers");

    /* Allocate output Y */
    sptStartTimer(timer);
    ind_buf = new sptIndex[X->nmodes * sizeof *ind_buf];
    spt_CheckOSError(!ind_buf, "Cuda SpTns * Vec");
    for(sptIndex m = 0; m < X->nmodes; ++m) {
        if(m < mode)
            ind_buf[m] = X->ndims[m];
        else if(m > mode)
            ind_buf[m - 1] = X->ndims[m];
    }
    result = sptNewSparseTensorWithNnz(Y, X->nmodes - 1, ind_buf, fiberidx.len - 1);
    spt_CheckError(result, "Cuda SpTns * Vec", NULL);
    free(ind_buf);
    sptStopTimer(timer);
    allocate_time = sptPrintElapsedTime(timer, "sptNewSparseTensorWithNnz");

    preprocess_time = sort_time + setfiber_time + allocate_time;
    printf("[Total preprocess time]: %lf\n", preprocess_time);

    /* Set indices */
    sptStartTimer(timer);
    sptSparseTensorSetIndices(Y, &fiberidx, mode, X);
    sptStopTimer(timer);
    copy_time_cpu = sptPrintElapsedTime(timer, "Copy indices");

    sptValue *Y_val = NULL;
    result = hipMalloc((void **) &Y_val, Y->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    sptValue *X_val = NULL;
    result = hipMalloc((void **) &X_val, X->nnz * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    sptIndex *X_inds_m = NULL;
    result = hipMalloc((void **) &X_inds_m, X->nnz * sizeof (sptIndex));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    sptValue *V_val = NULL;
    result = hipMalloc((void **) &V_val, V->len * sizeof (sptValue));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    sptNnzIndex *fiberidx_val = NULL;
    result = hipMalloc((void **) &fiberidx_val, fiberidx.len * sizeof (sptNnzIndex));
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");

    /* Copy data to GPU */
    sptStartTimer(timer);
    hipMemset(Y_val, 0, Y->nnz * sizeof (sptValue));
    hipMemcpy(X_val, X->values.data, X->nnz * sizeof (sptValue), hipMemcpyHostToDevice);
    hipMemcpy(X_inds_m, X->inds[mode].data, X->nnz * sizeof (sptIndex), hipMemcpyHostToDevice);
    hipMemcpy(V_val, V->data, V->len * sizeof (sptValue), hipMemcpyHostToDevice);
    hipMemcpy(fiberidx_val, fiberidx.data, fiberidx.len * sizeof (sptNnzIndex), hipMemcpyHostToDevice);
    sptStopTimer(timer);
    copy_time_gpu = sptPrintElapsedTime(timer, "Device copy");


    const sptNnzIndex max_nblocks = 32768;
    const sptNnzIndex max_nthreads_per_block = 256;

    sptNnzIndex nthreadsx = 1;
    sptNnzIndex nthreadsy = 1;
    sptNnzIndex all_nblocks = 0;
    sptNnzIndex nblocks = 0;

    const char *env_PASTA_TTV_NTHREADS = getenv("PASTA_TTV_NTHREADS");

    switch(impl_num) {
    // case 1:
    case 11: // Naive, 1D
        if(Y->nnz < max_nthreads_per_block) {
            nthreadsx = Y->nnz;
            nblocks = 1;
        } else {
            nthreadsx = max_nthreads_per_block;
            all_nblocks = (Y->nnz + nthreadsx -1) / nthreadsx;
            if(all_nblocks < max_nblocks) {
                nblocks = all_nblocks;
            } else {
                nblocks = max_nblocks;
            }
        }
        break;
    }
    dim3 dimBlock(nthreadsx, nthreadsy);
    printf("all_nblocks: %lu, nthreadsx: %lu, nthreadsy: %lu\n", all_nblocks, nthreadsx, nthreadsy);

    /* Computation */
    sptStartTimer(timer);

    switch(impl_num) {
    // case 1:
    case 11: // Naive
        printf("[Cuda SpTns * Vec] spt_TTVNnzKernel<<<%lu, (%lu, %lu)>>>\n", nblocks, nthreadsx, nthreadsy);
        spt_TTVNnzKernel<<<nblocks, dimBlock>>>(
            Y_val, Y->nnz,
            X_val, X->nnz, X_inds_m,
            fiberidx_val, fiberidx.len,
            V_val, V->len);
        break;
    }
    result = hipDeviceSynchronize();
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec kernel");

    sptStopTimer(timer);
    comp_time = sptPrintElapsedTime(timer, "Cuda SpTns * Vec");

    /* Copy back to CPU */
    sptStartTimer(timer);
    hipMemcpy(Y->values.data, Y_val, Y->nnz * sizeof (sptValue), hipMemcpyDeviceToHost);
    sptStopTimer(timer);
    copy_time_gpu += sptPrintElapsedTime(timer, "Device copy back");
    
    sptFreeTimer(timer);
    result = hipFree(fiberidx_val);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    result = hipFree(V_val);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    result = hipFree(X_inds_m);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    result = hipFree(X_val);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    result = hipFree(Y_val);
    spt_CheckCudaError(result != 0, "Cuda SpTns * Vec");
    sptFreeNnzIndexVector(&fiberidx);

    total_time = copy_time_cpu + copy_time_gpu + comp_time;
    printf("[Total time]: %lf\n", total_time);
    printf("\n");

    return 0;
}
